#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "cudart_platform.h"

#include <algorithm>
#include <iostream>
#include <cstdlib>

using uint8_t = unsigned char;

struct Vec2
{
	float x = 0.0, y = 0.0;

	__device__ Vec2 operator-(Vec2 other)
	{
		Vec2 res;
		res.x = this->x - other.x;
		res.y = this->y - other.y;
		return res;
	}

	__device__ Vec2 operator+(Vec2 other)
	{
		Vec2 res;
		res.x = this->x + other.x;
		res.y = this->y + other.y;
		return res;
	}

	__device__ Vec2 operator*(float d)
	{
		Vec2 res;
		res.x = this->x * d;
		res.y = this->y * d;
		return res;
	}
};

struct Color3f
{
	float R = 0.0f;
	float G = 0.0f;
	float B = 0.0f;

	__host__ __device__ Color3f operator+ (Color3f other)
	{
		Color3f res;
		res.R = this->R + other.R;
		res.G = this->G + other.G;
		res.B = this->B + other.B;
		return res;
	}

	__host__ __device__ Color3f operator* (float d)
	{
		Color3f res;
		res.R = this->R * d;
		res.G = this->G * d;
		res.B = this->B * d;
		return res;
	}
};

struct Particle
{
	Vec2 u; // velocity
	Color3f color;
};

static struct Config
{
	float velocityDiffusion;
	float pressure;
	float vorticity;
	float colorDiffusion;
	float densityDiffusion;
	float forceScale;
	float bloomIntense;
	int radius;
	bool bloomEnabled;
} config;

static struct SystemConfig
{
	int velocityIterations = 20;
	int pressureIterations = 40;
	int xThreads = 80;
	int yThreads = 1;
} sConfig;

void setConfig(
	float vDiffusion = 0.8f,
	float pressure = 1.5f,
	float vorticity = 50.0f,
	float cDiffuion = 0.8f,
	float dDiffuion = 1.2f,
	float force = 5000.0f,
	float bloomIntense = 0.1f,
	int radius = 400,
	bool bloom = true
)
{
	config.velocityDiffusion = vDiffusion;
	config.pressure = pressure;
	config.vorticity = vorticity;
	config.colorDiffusion = cDiffuion;
	config.densityDiffusion = dDiffuion;
	config.forceScale = force;
	config.bloomIntense = bloomIntense;
	config.radius = radius;
	config.bloomEnabled = bloom;
}

static const int colorArraySize = 7;
Color3f colorArray[colorArraySize];

static Particle* newField;
static Particle* oldField;
static uint8_t* colorField;
static size_t xSize, ySize;
static float* pressureOld;
static float* pressureNew;
static float* vorticityField;
static Color3f currentColor;
static float elapsedTime = 0.0f;
static float timeSincePress = 0.0f;

// inits all buffers, must be called before computeField function call
void cudaInit(size_t x, size_t y)
{
	setConfig();

	colorArray[0] = { 1.0f, 0.0f, 0.0f };
	colorArray[1] = { 0.0f, 1.0f, 0.0f };
	colorArray[2] = { 1.0f, 0.0f, 1.0f };
	colorArray[3] = { 1.0f, 1.0f, 0.0f };
	colorArray[4] = { 0.0f, 1.0f, 1.0f };
	colorArray[5] = { 1.0f, 0.0f, 1.0f };
	colorArray[6] = { 1.0f, 0.5f, 0.3f };

	int idx = rand() % colorArraySize;
	currentColor = colorArray[idx];

	xSize = x, ySize = y;

	hipSetDevice(0);
	hipMalloc(&colorField, xSize * ySize * 4 * sizeof(uint8_t));
	hipMalloc(&oldField, xSize * ySize * sizeof(Particle));
	hipMalloc(&newField, xSize * ySize * sizeof(Particle));
	hipMalloc(&pressureOld, xSize * ySize * sizeof(float));
	hipMalloc(&pressureNew, xSize * ySize * sizeof(float));
	hipMalloc(&vorticityField, xSize * ySize * sizeof(float));
}

// releases all buffers, must be called on program exit
void cudaExit()
{
	hipFree(colorField);
	hipFree(oldField);
	hipFree(newField);
	hipFree(pressureOld);
	hipFree(pressureNew);
	hipFree(vorticityField);
}

// interpolates quantity of grid cells
__device__ Particle interpolate(Vec2 v, Particle* field, size_t xSize, size_t ySize)
{
	float x1 = (int)v.x;
	float y1 = (int)v.y;
	float x2 = (int)v.x + 1;
	float y2 = (int)v.y + 1;
	Particle q1, q2, q3, q4;
	#define CLAMP(val, minv, maxv) min(maxv, max(minv, val))
	#define SET(Q, x, y) Q = field[int(CLAMP(y, 0.0f, ySize - 1.0f)) * xSize + int(CLAMP(x, 0.0f, xSize - 1.0f))]	
	SET(q1, x1, y1);
	SET(q2, x1, y2);
	SET(q3, x2, y1);
	SET(q4, x2, y2);
	#undef SET
	#undef CLAMP
	float t1 = (x2 - v.x) / (x2 - x1);
	float t2 = (v.x - x1) / (x2 - x1);
	Vec2 f1 = q1.u * t1 + q3.u * t2;
	Vec2 f2 = q2.u * t1 + q4.u * t2;
	Color3f C1 = q2.color * t1 + q4.color * t2;
	Color3f C2 = q2.color * t1 + q4.color * t2;
	float t3 = (y2 - v.y) / (y2 - y1);
	float t4 = (v.y - y1) / (y2 - y1);
	Particle res;
	res.u = f1 * t3 + f2 * t4;
	res.color = C1 * t3 + C2 * t4;
	return res;
}

// performs iteration of jacobi method on velocity grid field
__device__ Vec2 jacobiVelocity(Particle* field, size_t xSize, size_t ySize, Vec2 v, Vec2 B, float alpha, float beta)
{
	Vec2 vU = B * -1.0f, vD = B * -1.0f, vR = B * -1.0f, vL = B * -1.0f;
	#define SET(U, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) U = field[int(y) * xSize + int(x)].u
	SET(vU, v.x, v.y - 1);
	SET(vD, v.x, v.y + 1);
	SET(vL, v.x - 1, v.y);
	SET(vR, v.x + 1, v.y);
	#undef SET
	v = (vU + vD + vL + vR + B * alpha) * (1.0f / beta);
	return v;
}

// performs iteration of jacobi method on pressure grid field
__device__ float jacobiPressure(float* pressureField, size_t xSize, size_t ySize, int x, int y, float B, float alpha, float beta)
{
	float C = pressureField[int(y) * xSize + int(x)];
	float xU = C, xD = C, xL = C, xR = C;
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = pressureField[int(y) * xSize + int(x)]
	SET(xU, x, y - 1);
	SET(xD, x, y + 1);
	SET(xL, x - 1, y);
	SET(xR, x + 1, y);
	#undef SET
	float pressure = (xU + xD + xL + xR + alpha * B) * (1.0f / beta);
	return pressure;
}

// performs iteration of jacobi method on color grid field
__device__ Color3f jacobiColor(Particle* colorField, size_t xSize, size_t ySize, Vec2 pos, Color3f B, float alpha, float beta)
{
	Color3f xU, xD, xL, xR, res;
	int x = pos.x;
	int y = pos.y;
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = colorField[int(y) * xSize + int(x)]
	SET(xU, x, y - 1).color;
	SET(xD, x, y + 1).color;
	SET(xL, x - 1, y).color;
	SET(xR, x + 1, y).color;
	#undef SET
	res = (xU + xD + xL + xR + B * alpha) * (1.0f / beta);
	return res;
}

// computes divergency of velocity field
__device__ float divergency(Particle* field, size_t xSize, size_t ySize, int x, int y)
{
	Particle& C = field[int(y) * xSize + int(x)];
	float x1 = -1 * C.u.x, x2 = -1 * C.u.x, y1 = -1 * C.u.y, y2 = -1 * C.u.y;
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = field[int(y) * xSize + int(x)]
	SET(x1, x + 1, y).u.x;
	SET(x2, x - 1, y).u.x;
	SET(y1, x, y + 1).u.y;
	SET(y2, x, y - 1).u.y;
	#undef SET
	return (x1 - x2 + y1 - y2) * 0.5f;
}

// computes gradient of pressure field
__device__ Vec2 gradient(float* field, size_t xSize, size_t ySize, int x, int y)
{
	float C = field[y * xSize + x];
#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = field[int(y) * xSize + int(x)]
	float x1 = C, x2 = C, y1 = C, y2 = C;
	SET(x1, x + 1, y);
	SET(x2, x - 1, y);
	SET(y1, x, y + 1);
	SET(y2, x, y - 1);
#undef SET
	Vec2 res = { (x1 - x2) * 0.5f, (y1 - y2) * 0.5f };
	return res;
}

// computes absolute value gradient of vorticity field
__device__ Vec2 absGradient(float* field, size_t xSize, size_t ySize, int x, int y)
{
	float C = field[int(y) * xSize + int(x)];
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = field[int(y) * xSize + int(x)]
	float x1 = C, x2 = C, y1 = C, y2 = C;
	SET(x1, x + 1, y);
	SET(x2, x - 1, y);
	SET(y1, x, y + 1);
	SET(y2, x, y - 1);
	#undef SET
	Vec2 res = { (abs(x1) - abs(x2)) * 0.5f, (abs(y1) - abs(y2)) * 0.5f };
	return res;
}

// computes curl of velocity field
__device__ float curl(Particle* field, size_t xSize, size_t ySize, int x, int y)
{
	Vec2 C = field[int(y) * xSize + int(x)].u;
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = field[int(y) * xSize + int(x)]
	float x1 = -C.x, x2 = -C.x, y1 = -C.y, y2 = -C.y;
	SET(x1, x + 1, y).u.x;
	SET(x2, x - 1, y).u.x;
	SET(y1, x, y + 1).u.y;
	SET(y2, x, y - 1).u.y;
	#undef SET
	float res = ((y1 - y2) - (x1 - x2)) * 0.5f;
	return res;
}

// adds quantity to particles using bilinear interpolation
__global__ void advect(Particle* newField, Particle* oldField, size_t xSize, size_t ySize, float dDiffusion, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float decay = 1.0f / (1.0f + dDiffusion * dt);
	Vec2 pos = { x * 1.0f, y * 1.0f };
	Particle& Pold = oldField[y * xSize + x];
	// find new particle tracing where it came from
	Particle p = interpolate(pos - Pold.u * dt, oldField, xSize, ySize);
	p.u = p.u * decay;
	p.color.R = min(1.0f, pow(p.color.R, 1.005f) * decay);
	p.color.G = min(1.0f, pow(p.color.G, 1.005f) * decay);
	p.color.B = min(1.0f, pow(p.color.B, 1.005f) * decay);
	newField[y * xSize + x] = p;
}

// calculates color field diffusion
__global__ void computeColor(Particle* newField, Particle* oldField, size_t xSize, size_t ySize, float cDiffusion, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	Vec2 pos = { x * 1.0f, y * 1.0f };
	Color3f c = oldField[y * xSize + x].color;
	float alpha = cDiffusion * cDiffusion / dt;
	float beta = 4.0f + alpha;
	// perfom one iteration of jacobi method (diffuse method should be called 20-50 times per cell)
	newField[y * xSize + x].color = jacobiColor(oldField, xSize, ySize, pos, c, alpha, beta);
}

// fills output image with corresponding color
__global__ void paint(uint8_t* colorField, Particle* field, size_t xSize, size_t ySize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	float R = field[y * xSize + x].color.R;
	float G = field[y * xSize + x].color.G;
	float B = field[y * xSize + x].color.B;

	colorField[4 * (y * xSize + x) + 0] = min(255.0f, 255.0f * R);
	colorField[4 * (y * xSize + x) + 1] = min(255.0f, 255.0f * G);
	colorField[4 * (y * xSize + x) + 2] = min(255.0f, 255.0f * B);
	colorField[4 * (y * xSize + x) + 3] = 255;
}

// calculates nonzero divergency velocity field u
__global__ void diffuse(Particle* newField, Particle* oldField, size_t xSize, size_t ySize, float vDiffusion, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	Vec2 pos = { x * 1.0f, y * 1.0f };
	Vec2 u = oldField[y * xSize + x].u;
	// perfoms one iteration of jacobi method (diffuse method should be called 20-50 times per cell)
	float alpha = vDiffusion * vDiffusion / dt;
	float beta = 4.0f + alpha;
	newField[y * xSize + x].u = jacobiVelocity(oldField, xSize, ySize, pos, u, alpha, beta);
}

// performs iteration of jacobi method on pressure field
__global__ void computePressureImpl(Particle* field, size_t xSize, size_t ySize, float* pNew, float* pOld, float pressure, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float div = divergency(field, xSize, ySize, x, y);

	float alpha = -1.0f * pressure * pressure;
	float beta = 4.0;
	pNew[y * xSize + x] = jacobiPressure(pOld, xSize, ySize, x, y, div, alpha, beta);
}

// projects pressure field on velocity field
__global__ void project(Particle* newField, size_t xSize, size_t ySize, float* pField)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	Vec2& u = newField[y * xSize + x].u;
	u = u - gradient(pField, xSize, ySize, x, y);
}

// applies force and add color dye to the particle field
__global__ void applyForce(Particle* field, size_t xSize, size_t ySize, Color3f color, Vec2 F, Vec2 pos, int r, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float e = expf((-(powf(x - pos.x, 2) + powf(y - pos.y, 2))) / r);
	Vec2 uF = F * dt * e;
	Particle& p = field[y * xSize + x];
	p.u = p.u + uF;
	color = color * e + p.color;
	p.color.R = min(1.0f, color.R);
	p.color.G = min(1.0f, color.G);
	p.color.B = min(1.0f, color.B);
}

// computes vorticity field which should be passed to applyVorticity function
__global__ void computeVorticity(float* vField, Particle* field, size_t xSize, size_t ySize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	vField[y * xSize + x] = curl(field, xSize, ySize, x, y);
}

// applies vorticity to velocity field
__global__ void applyVorticity(Particle* newField, Particle* oldField, float* vField, size_t xSize, size_t ySize, float vorticity, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	Particle& pOld = oldField[y * xSize + x];
	Particle& pNew = newField[y * xSize + x];

	Vec2 v = absGradient(vField, xSize, ySize, x, y);
	v.y *= -1.0f;

	float length = sqrtf(v.x * v.x + v.y * v.y) + 1e-5f;
	Vec2 vNorm = v * (1.0f / length);

	Vec2 vF = vNorm * vField[y * xSize + x] * vorticity;
	pNew = pOld;
	pNew.u = pNew.u + vF * dt;
}

// adds flashlight effect near the mouse position
__global__ void applyBloom(uint8_t* colorField, size_t xSize, size_t ySize, int xpos, int ypos, float radius, float bloomIntense)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pos = 4 * (y * xSize + x);

	float e = bloomIntense * expf(-(powf(x - xpos, 2) + powf(y - ypos, 2)) / pow(radius, 2));

	uint8_t R = colorField[pos + 0];
	uint8_t G = colorField[pos + 1];
	uint8_t B = colorField[pos + 2];

	uint8_t maxval = max(R, max(G, B));

	colorField[pos + 0] = min(255.0f, R + maxval * e);
	colorField[pos + 1] = min(255.0f, G + maxval * e);
	colorField[pos + 2] = min(255.0f, B + maxval * e);
}

// performs several iterations over velocity and color fields
void computeDiffusion(dim3 numBlocks, dim3 threadsPerBlock, float dt)
{
	// diffuse velocity and color
	for (int i = 0; i < sConfig.velocityIterations; i++)
	{
		diffuse<<<numBlocks, threadsPerBlock>>>(newField, oldField, xSize, ySize, config.velocityDiffusion, dt);
		computeColor<<<numBlocks, threadsPerBlock>>>(newField, oldField, xSize, ySize, config.colorDiffusion, dt);
		std::swap(newField, oldField);
	}
}

// performs several iterations over pressure field
void computePressure(dim3 numBlocks, dim3 threadsPerBlock, float dt)
{
	for (int i = 0; i < sConfig.pressureIterations; i++)
	{
		computePressureImpl<<<numBlocks, threadsPerBlock>>>(oldField, xSize, ySize, pressureNew, pressureOld, config.pressure, dt);
		std::swap(pressureOld, pressureNew);
	}
}

// main function, calls vorticity -> diffusion -> force -> pressure -> project -> advect -> paint -> bloom
void computeField(uint8_t* result, float dt, int x1pos, int y1pos, int x2pos, int y2pos, bool isPressed)
{
	dim3 threadsPerBlock(sConfig.xThreads, sConfig.yThreads);
	dim3 numBlocks(xSize / threadsPerBlock.x, ySize / threadsPerBlock.y);

	// curls and vortisity
	computeVorticity<<<numBlocks, threadsPerBlock>>>(vorticityField, oldField, xSize, ySize);
	applyVorticity<<<numBlocks, threadsPerBlock>>>(newField, oldField, vorticityField, xSize, ySize, config.vorticity, dt);
	std::swap(oldField, newField);

	// diffuse velocity and color
	computeDiffusion(numBlocks, threadsPerBlock, dt);

	// apply force
	if (isPressed)
	{
		timeSincePress = 0.0f;
		elapsedTime += dt;
		// apply gradient to color
		int roundT = int(elapsedTime) % colorArraySize;
		int ceilT = int((elapsedTime) + 1) % colorArraySize;
		float w = elapsedTime - int(elapsedTime);
		currentColor = colorArray[roundT] * (1 - w) + colorArray[ceilT] * w;

		Vec2 F;
		float scale = config.forceScale;
		F.x = (x2pos - x1pos) * scale;
		F.y = (y2pos - y1pos) * scale;
		Vec2 pos = { x2pos * 1.0f, y2pos * 1.0f };
		applyForce<<<numBlocks, threadsPerBlock>>>(oldField, xSize, ySize, currentColor, F, pos, config.radius, dt);
	}
	else
	{
		timeSincePress += dt;
	}

	// compute pressure
	computePressure(numBlocks, threadsPerBlock, dt);

	// project
	project<<<numBlocks, threadsPerBlock>>>(oldField, xSize, ySize, pressureOld);
	hipMemset(pressureOld, 0, xSize * ySize * sizeof(float));

	// advect
	advect<<<numBlocks, threadsPerBlock>>>(newField, oldField, xSize, ySize, config.densityDiffusion, dt);
	std::swap(newField, oldField);

	// paint image
	paint<<<numBlocks, threadsPerBlock>>>(colorField, oldField, xSize, ySize);

	// apply bloom in mouse pos
	if (config.bloomEnabled && timeSincePress < 5.0f)
	{
		applyBloom<<<numBlocks, threadsPerBlock>>>(colorField, xSize, ySize, x2pos, y2pos, config.radius, config.bloomIntense);
	}

	// copy image to cpu
	size_t size = xSize * ySize * 4 * sizeof(uint8_t);
	hipMemcpy(result, colorField, size, hipMemcpyDeviceToHost);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		std::cout << hipGetErrorName(error) << std::endl;
	}
}